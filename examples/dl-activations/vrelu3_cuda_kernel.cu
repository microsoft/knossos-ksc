#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

template<typename scalar_t> using tensor_accessor_1 =
    torch::PackedTensorAccessor32<scalar_t,1,torch::RestrictPtrTraits>;
template<typename scalar_t> using tensor_accessor_2 =
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits>;

template <typename scalar_t>
inline __device__ scalar_t relu3_forward(scalar_t input) {
  if (input < (scalar_t)0.0) {
    return (scalar_t)0.0;
  } else if (input < (scalar_t)1.0) {
    return (scalar_t)1/3 * input * input * input;
  } else {
    return input - (scalar_t)2/3;
  }
}

template <typename scalar_t>
__global__ void vrelu3_cuda_forward_kernel_1(
    const tensor_accessor_1<scalar_t> input,
    tensor_accessor_1<scalar_t> output) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < input.size(0))
    output[i] = relu3_forward(input[i]);
}

template <typename scalar_t>
__global__ void vrelu3_cuda_forward_kernel_2(
    const tensor_accessor_2<scalar_t> input,
    tensor_accessor_2<scalar_t> output) {
  const int n = blockIdx.y;
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < input.size(1))
    output[n][i] = relu3_forward(input[n][i]);
}

torch::Tensor vrelu3_cuda_forward(torch::Tensor input) {
  auto output = torch::zeros_like(input);

  switch (input.sizes().size()) {
    case 1: {
      const auto input_size = input.size(0);

      // TODO: find out how PyTorch chooses these parameters
      const int threads = 1024;
      const int blocks = (input_size + threads - 1) / threads;

      AT_DISPATCH_FLOATING_TYPES(input.type(), "vrelu3_forward_cuda (rank 1)", ([&] {
        vrelu3_cuda_forward_kernel_1<scalar_t><<<blocks, threads>>>(
            input.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>(),
            output.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>());
      }));
      break;
    }
    case 2: {
      const auto input_size_0 = input.size(0);
      const auto input_size_1 = input.size(1);

      const int threads = 1024;
      const dim3 blocks((input_size_1 + threads - 1) / threads, input_size_0);

      AT_DISPATCH_FLOATING_TYPES(input.type(), "vrelu3_forward_cuda (rank 2)", ([&] {
        vrelu3_cuda_forward_kernel_2<scalar_t><<<blocks, threads>>>(
            input.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
            output.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>());
      }));
      break;
    }
    default:
      TORCH_CHECK(false, "Unsupported tensor rank");
  }
  return output;
}

template <typename scalar_t>
inline __device__ scalar_t relu3_backward(scalar_t grad, scalar_t x) {
  if (x < (scalar_t)0.0) {
    return (scalar_t)0.0;
  } else if (x < (scalar_t)1.0) {
    return x * x * grad;
  } else {
    return grad;
  }
}

template <typename scalar_t>
__global__ void vrelu3_cuda_backward_kernel_1(
    tensor_accessor_1<scalar_t> d_x,
    const tensor_accessor_1<scalar_t> grad,
    const tensor_accessor_1<scalar_t> x) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < x.size(0))
    d_x[i] = relu3_backward(grad[i], x[i]);
}

template <typename scalar_t>
__global__ void vrelu3_cuda_backward_kernel_2(
    tensor_accessor_2<scalar_t> d_x,
    const tensor_accessor_2<scalar_t> grad,
    const tensor_accessor_2<scalar_t> x) {
  const int n = blockIdx.y;
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < x.size(1))
    d_x[n][i] = relu3_backward(grad[n][i], x[n][i]);
}

torch::Tensor vrelu3_cuda_backward(
    torch::Tensor grad,
    torch::Tensor x) {
  auto d_x = torch::zeros_like(x);
  switch (x.sizes().size()) {
    case 1: {
      auto x_size = x.size(0);

      // TODO: find out how PyTorch chooses these parameters
      const int threads = 1024;
      const int blocks = (x_size + threads - 1) / threads;

      AT_DISPATCH_FLOATING_TYPES(x.type(), "vrelu3_backward_cuda (rank 1)", ([&] {
        vrelu3_cuda_backward_kernel_1<scalar_t><<<blocks, threads>>>(
            d_x.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>(),
            grad.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>(),
            x.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>());
      }));
      break;
    }
    case 2: {
      auto x_size_0 = x.size(0);
      auto x_size_1 = x.size(1);

      const int threads = 1024;
      const dim3 blocks((x_size_1 + threads - 1) / threads, x_size_0);

      AT_DISPATCH_FLOATING_TYPES(x.type(), "vrelu3_backward_cuda (rank 2)", ([&] {
        vrelu3_cuda_backward_kernel_2<scalar_t><<<blocks, threads>>>(
            d_x.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
            grad.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
            x.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>());
      }));
      break;
    }
    default:
      TORCH_CHECK(false, "Unsupported tensor rank");
  }
  return d_x;
}


