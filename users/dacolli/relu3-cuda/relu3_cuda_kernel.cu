#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

template <typename scalar_t>
__global__ void relu3_cuda_forward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,1,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,1,torch::RestrictPtrTraits> output) {
  // element index
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < input.size(0)){
    if (input[i] >= 0) {
      if (input[i] > 1) {
        output[i] = input[i] - (scalar_t)2/3;
      } else {
        output[i] = (scalar_t)1/3 * input[i] * input[i] * input[i];
      }
    }
  }
}

torch::Tensor relu3_cuda_forward(
    torch::Tensor input) {
  auto output = torch::zeros_like(input);
  const auto input_size = input.size(0);

  const int threads = 1024;
  const int blocks = (input_size + threads - 1) / threads;

  AT_DISPATCH_FLOATING_TYPES(input.type(), "relu3_forward_cuda", ([&] {
    relu3_cuda_forward_kernel<scalar_t><<<blocks, threads>>>(
        input.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>(),
        output.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>());
  }));

  return output;
}

template <typename scalar_t>
__global__ void relu3_cuda_backward_kernel(
    torch::PackedTensorAccessor32<scalar_t,1,torch::RestrictPtrTraits> d_x,
    const torch::PackedTensorAccessor32<scalar_t,1,torch::RestrictPtrTraits> grad,
    const torch::PackedTensorAccessor32<scalar_t,1,torch::RestrictPtrTraits> x) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < x.size(0)){
    if (x[i] >= 0) {
      if (x[i] > 1) {
        d_x[i] = grad[i];
      } else {
        d_x[i] = x[i] * x[i] * grad[i];
      }
    }
  }
}

torch::Tensor relu3_cuda_backward(
    torch::Tensor grad,
    torch::Tensor x) {
  auto d_x = torch::zeros_like(x);
  auto x_size = x.size(0);

  const int threads = 1024;
  const int blocks = (x_size + threads - 1) / threads;

  AT_DISPATCH_FLOATING_TYPES(x.type(), "relu3_backward_cuda", ([&] {
    relu3_cuda_backward_kernel<scalar_t><<<blocks, threads>>>(
        d_x.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>(),
        grad.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>(),
        x.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>());
  }));

  return d_x;
}


